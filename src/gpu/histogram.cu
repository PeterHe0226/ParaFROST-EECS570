/***********************************************************************[histogram.cu]
Copyright(c) 2020, Muhammad Osama - Anton Wijs,
Copyright(c) 2022-present, Muhammad Osama.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**********************************************************************************/

#include "solver.hpp"
using namespace ParaFROST;

uint32* Solver::flattenCNF(const uint32& numLits)
{
	assert(numLits);
	uint32* literals = cumm.resizeLits(numLits);
	if (flattened || !literals) return literals;
	LOGN2(2, " Copying survived literals..");
	copyIfAsync(literals, cnf);
	LOGENDING(2, 5, "(%d copied)", numLits);
	flattened = true;
	return literals;
}

void Solver::histSimp(const uint32& numLits)
{
	LOGN2(2, " Computing histogram on %d elements..", numLits);
	assert(numLits);
	cuLits& culits = cumm.literals();
	assert(culits.size >= numLits);
	t_iptr& thrust_lits = culits.thrust_lits;
	t_iptr& thrust_hist = cuhist.thrust_hist;
	SYNC(0); // sync 'flattenCNF'
	if (gopts.profile_gpu) cutimer->start();
	cacher.insert(cumm.scatter(), cumm.scatterCap());
	thrust::sort(thrust::cuda::par(tca), thrust_lits, thrust_lits + numLits);
	thrust::counting_iterator<size_t> search_begin(0);
	thrust::upper_bound(thrust::cuda::par(tca), thrust_lits, thrust_lits + numLits, search_begin, search_begin + inf.nDualVars, thrust_hist);
	thrust::adjacent_difference(thrust::cuda::par(tca), thrust_hist, thrust_hist + inf.nDualVars, thrust_hist);
	cacher.erase(cumm.scatterCap());
	if (gopts.profile_gpu) cutimer->stop(), cutimer->vo += cutimer->gpuTime();
	LOGDONE(2, 5);
}